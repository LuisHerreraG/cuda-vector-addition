
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline void checkCuda(hipError_t error)
{
  if (error != hipSuccess)
  {
    printf("Cuda error: %s\n", hipGetErrorString(error));
    assert(error == hipSuccess);
  }
}


void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  addVectorsInto<<<32,32>>>(c, a, b, N);

  checkCuda(hipGetLastError());
  checkCuda(hipDeviceSynchronize());

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
